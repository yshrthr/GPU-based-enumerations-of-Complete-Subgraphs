#include "hip/hip_runtime.h"
#include<iostream>
#include<cstring>
#include<hip/hip_runtime.h>
#include<fstream>
#include<thrust/device_vector.h>

#define BLOCKSIZE 512

using namespace std;

//Graph 
  bool *adjacencyMatrix;

__global__ void kclique(int* d_degree, int presentNodes, int K, int N,bool* adjacencyMat,int *d_count)
{
    
    int id = blockIdx.x*blockDim.x + threadIdx.x;
    
    if(id < N)
    {
        int st[10000][4];
        int top=-1;

        int dclique[10000];
       
        top++;
        
        dclique[1]=id+1;
     
        st[top][0]=id+2;
        st[top][1]=1;
        st[top][2]=2;
        st[top][3]=K;
        
        d_count[0]=0;
     
        while(top!=-1)
        {
            int j=st[top][0];
            int i=st[top][1];
            int l=st[top][2];
            int s=st[top][3];

            top--;
            if(j+1<=N)
            {
                top++;
             
                st[top][0]=j+1;
                st[top][1]=i;
                st[top][2]=l;
                st[top][3]=s;
            }
         
            if(d_degree[j]>=s-1)
            {
                dclique[l]=j;

                //check if the vertices form a clique or not
                bool flag=true;
             

                //Run a loop for all the set of edges for a vertex
                for(int x=1;x<l+1;x++)
                {
                    for(int y=x+1;y<l+1;y++)
                    {
                        if(adjacencyMat[ 1ll*dclique[x]*1000000+dclique[y] ]==false)
                        {    
                            flag=false;
                            break;
                        }
                    }
                    if(!flag)
                        break;
                }

                //flag will be true if it is clique
                if(flag)
                {
                    if(l<s)
                    {
                        top++;
                     
                        st[top][0]=j+1;
                        st[top][1]=j+1;
                        st[top][2]=l+1;
                        st[top][3]=s;
                    }
                 
                 //If it is clique then atomically increase the count array
                    else
                    {
                       atomicInc((unsigned int*) &d_count[0], 1000000);
                    }
                }
            }
        }
    }
    __syncthreads();
}

int main(int argc, char *argv[])
{
    int k;

    //Degree of the vertices
    int *degree;

    string path = argv[1];
    cin>>k;
    
    //path = "/content/drive/MyDrive/GPU_CP/graph.txt";
    //k = 3;

    degree = (int*)malloc(sizeof(int)*1000000);

    //adjacencyMatrix = (bool*)malloc(sizeof(bool)*1000000000000);
    //bool *d_adjacencyMatrix;

    hipMallocManaged(&adjacencyMatrix, 1000000000000*sizeof(bool));
    
    ifstream MyReadFile(path);
    string myText;
    int n=0;
    while (getline (MyReadFile, myText)){
        int a,b,i=0;
        string t="";
        while(myText[i]!=' ')
        {
            t+=myText[i];
            i++;
        }
        a=stoi(t);
        b=stoi(myText.substr(i+1));

        n = max(n,max(a,b));

        adjacencyMatrix[1ll*a*1000000+b] = true;
        adjacencyMatrix[1ll*b*1000000+a] = true;
        degree[a]++;
        degree[b]++;
    }



    int count[1];

    int *d_degree,*d_count;
    hipMalloc((void**)&d_degree, 10000*sizeof(int));
    hipMalloc((void**)&d_count, 1*sizeof(int));
    
    hipMemcpy(d_degree, degree, 10000*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_count, count, 1*sizeof(int), hipMemcpyHostToDevice);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float gpu_time = 0.0f;
    
    hipEventRecord(start, 0); 

    int blocks = 41;

    kclique<<<blocks,BLOCKSIZE>>>(d_degree,1,k,n,adjacencyMatrix,d_count);
    hipEventRecord(stop, 0); 
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&gpu_time, start, stop);
    
    hipMemcpy(count, d_count, 1*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    cout<<"Number of cliques of size "<<k<<" in the given graph are "<<count[0]<<endl;
    cout<<"Execution Time: "<<gpu_time<<" ms"<<endl;
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    hipFree(d_degree);
    hipFree(d_count);

    free(degree);

    return 0;
}
